#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <torch/extension.h>
#include "common.h"


inline __device__ void fetch2shared(
    int32_t n,
    const uint2 range,
    const uint *__restrict__ patchs_gs_id,
    const float *__restrict__ us,
    const float *__restrict__ cov2d_inv,
    const float *__restrict__ alphas,
    const float *__restrict__ colors,
    float2 *shared_pos2d,
    float3 *shared_cinv2d,
    float *shared_alpha,
    float3 *shared_color)
{
    int i = blockDim.x * threadIdx.y + threadIdx.x;  // block idx
    int j = range.x + n * BLOCK_SIZE + i;  // patch idx
    if (j < range.y)
    {
        int gs_id = patchs_gs_id[j];
        shared_pos2d[i].x = us[gs_id * 2];
        shared_pos2d[i].y = us[gs_id * 2 + 1];
        shared_cinv2d[i].x = cov2d_inv[gs_id * 3];
        shared_cinv2d[i].y = cov2d_inv[gs_id * 3 + 1];
        shared_cinv2d[i].z = cov2d_inv[gs_id * 3 + 2];
        shared_alpha[i] =   alphas[gs_id];
        shared_color[i].x = colors[gs_id * 3];
        shared_color[i].y = colors[gs_id * 3 + 1];
        shared_color[i].z = colors[gs_id * 3 + 2];
    }
}

__global__ void createKey(const int gs_num,
                          const dim3 grid,
                          const uint4 *__restrict__ rects,
                          const float *__restrict__ depths,
                          const uint *__restrict__ offsets,
                          uint64_t *__restrict__ patch_keys,
                          uint *__restrict__ patchs_gs_id)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= gs_num)
		return;
    uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
    uint4 rect = rects[idx];

		for (uint y = rect.y; y < rect.w; y++)
		{
			for (uint x = rect.x; x < rect.z; x++)
			{
				uint64_t key = (y * grid.x + x);
				key <<= 32;
                uint32_t depth_cm = depths[idx] * 1000; // mm
				key |= depth_cm;
				patch_keys[off] = key;
				patchs_gs_id[off] = idx;
				off++;
			}
		}
}

__global__ void getRect(
    const int W,
    const int H,
    int gs_num,
    const float *__restrict__ us,
    const float *__restrict__ areas,
    const float *__restrict__ depths,
    const dim3 grid,
    uint4 *__restrict__ gs_rects,
    uint *__restrict__ gs_patch_num)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= gs_num)
		return;

    float d = depths[idx];
    float2 u = {us[idx*2], us[idx*2 + 1]};

    float x_norm =  u.x / W * 2.f - 1.f;
    float y_norm =  u.y / H * 2.f - 1.f;
    if (abs(x_norm) > 1.3 || abs(y_norm) > 1.3 || d < 0.1 || d > 100)
    {
        gs_rects[idx] = {0, 0, 0, 0};
        gs_patch_num[idx] = 0;
		return;
    }

    float xs = areas[idx*2];
    float ys = areas[idx*2 + 1];

    uint4 rect = {
		min(grid.x, max((int)0, (int)((u.x - xs) / BLOCK))),  // min_x
		min(grid.y, max((int)0, (int)((u.y - ys) / BLOCK))),  // min_y
        min(grid.x, max((int)0, (int)(DIV_ROUND_UP(u.x + xs, BLOCK)))),  // max_x
		min(grid.y, max((int)0, (int)(DIV_ROUND_UP(u.y + ys, BLOCK))))   // max_y
	};

    gs_rects[idx] = rect;
    gs_patch_num[idx] = (rect.z - rect.x) * (rect.w - rect.y);
}

__global__ void getRange(
    const int patch_num,
    const uint64_t *__restrict__ patch_keys,
    uint2 *__restrict__ gs_ranges)
{
    const int cur_patch = blockIdx.x * blockDim.x + threadIdx.x;

    if (cur_patch >= patch_num)
        return;

    const int prv_patch = cur_patch == 0 ? 0 : cur_patch - 1;

    uint32_t cur_tile = patch_keys[cur_patch] >> 32;
    uint32_t prv_tile = patch_keys[prv_patch] >> 32;

    if (cur_patch == 0)
        gs_ranges[cur_tile].x = 0;
    else if (cur_patch == patch_num - 1)
        gs_ranges[cur_tile].y = patch_num;

    if (prv_tile != cur_tile)
    {
        gs_ranges[prv_tile].y = cur_patch;
        gs_ranges[cur_tile].x = cur_patch;
    }
}

__global__ void  draw __launch_bounds__(BLOCK * BLOCK)(
    const int W,
    const int H,
    const uint2 *__restrict__ gs_ranges,
    const uint *__restrict__ patchs_gs_id,
    const float *__restrict__ us,
    const float *__restrict__ cov2d_inv,
    const float *__restrict__ alphas,
    const float *__restrict__ colors,
    float *__restrict__ image)

{
    const uint2 tile = {blockIdx.x, blockIdx.y};
    const uint2 pix = {tile.x * BLOCK + threadIdx.x,
                       tile.y * BLOCK + threadIdx.y};

    const int tile_idx = tile.y * gridDim.x + tile.x;
    const uint32_t pix_idx = W * pix.y + pix.x;

	const bool inside = pix.x < W && pix.y < H;
	const uint2 range = gs_ranges[tile_idx];


	bool thread_is_finished = !inside;

	__shared__ float2 shared_pos2d[BLOCK_SIZE];
	__shared__ float3 shared_cinv2d[BLOCK_SIZE];
    __shared__ float  shared_alpha[BLOCK_SIZE];
    __shared__ float3 shared_color[BLOCK_SIZE];

	const int gs_num = range.y - range.x;

    float3 finial_color = {0, 0, 0};

    float tau = 1.0f;

    // for all 2d gaussian 
    for (int i = 0; i < gs_num; i++)
    {
        int finished_thread_num = __syncthreads_count(thread_is_finished);

        if (finished_thread_num == BLOCK_SIZE)
            break;

        int j = i % BLOCK_SIZE;

        if (j == 0)
        {
            // fetch 2d gaussian data to share memory
            fetch2shared(i / BLOCK_SIZE,
                         range,
                         patchs_gs_id,
                         us,
                         cov2d_inv,
                         alphas,
                         colors,
                         shared_pos2d,
                         shared_cinv2d,
                         shared_alpha,
                         shared_color);
            __syncthreads();
        }

        // get 2d gaussian info for current tile (pix share the same info within the tile)
        float2 u = shared_pos2d[j];
        float3 cinv = shared_cinv2d[j];
        float alpha = shared_alpha[j];
        float3 color = shared_color[j];
        float2 d = u - pix;

        // forward.md (5.1)
        // mahalanobis squared distance for 2d gaussian to this pix
        float maha_dist = max(0.0f,  mahaSqDist(cinv, d));

        float alpha_prime = min(0.99f, alpha * exp( -0.5f * maha_dist));

        if (alpha_prime < 0.002f)
            continue;

        // forward.md (5)
        finial_color +=  tau * alpha_prime * color;

        // forward.md (5.2)
        float tau_new = tau * (1.f - alpha_prime);

        if (tau_new < 0.0001f)
        {
            thread_is_finished = true;
            continue;
        }
        tau = tau_new;
    }

    if (inside)
    {
        image[H * W * 0 + pix_idx] = finial_color.x;
        image[H * W * 1 + pix_idx] = finial_color.y;
        image[H * W * 2 + pix_idx] = finial_color.z;
    }
}

__global__ void inverseCov2D(
    int gs_num,
    const float *__restrict__ cov2d,
    float *__restrict__ cov2d_inv,
    float *__restrict__ areas)
{
    // compute inverse of cov2d
    // Determine the drawing area of 2d Gaussian.

    const int gs_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (gs_id >= gs_num)
		return;
    // forward.md 5.3
    const float a = cov2d[gs_id * 3];
    const float b = cov2d[gs_id * 3 + 1];
    const float c = cov2d[gs_id * 3 + 2];

    const float det_inv = 1./(a*c - b*b);
    cov2d_inv[gs_id * 3 + 0] =  det_inv * c;
    cov2d_inv[gs_id * 3 + 1] = -det_inv * b;
    cov2d_inv[gs_id * 3 + 2] =  det_inv * a;
    areas[gs_id * 2 + 0] =  3 * sqrt(a);
    areas[gs_id * 2 + 1] =  3 * sqrt(c);
}

std::vector<torch::Tensor> rasterizGuassian2DCUDA(
    torch::Tensor us,
    torch::Tensor cov2d,
    torch::Tensor alphas,
    torch::Tensor depths,
    torch::Tensor colors,
    int H,
    int W)
{
    auto float_opts = us.options().dtype(torch::kFloat32);
    torch::Tensor image = torch::full({3, H, W}, 0.0, float_opts);

    //gs:    2d gaussian;  a projection of a 3d gaussian onto a 2d image
    //tile:  a 16x16 area of 2d image
    //patch: a 2d gaussian may cover on many tiles, a 2d gaussian on a tile is called a patch

    // the total number of 2d gaussian.
    int gs_num = us.sizes()[0]; 
    
    dim3 grid(DIV_ROUND_UP(W, BLOCK), DIV_ROUND_UP(H, BLOCK), 1);
	dim3 block(BLOCK, BLOCK, 1);

    thrust::device_vector<uint4> gs_rects(gs_num);
    thrust::device_vector<uint>  gs_patch_num(gs_num);
    thrust::device_vector<uint>  gs_patch_offsets(gs_num);
    thrust::device_vector<float>  cov2d_inv(gs_num * 3);
    thrust::device_vector<float>  areas(gs_num * 2);

    inverseCov2D<<<DIV_ROUND_UP(gs_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        gs_num,
        cov2d.contiguous().data<float>(),
        thrust::raw_pointer_cast(cov2d_inv.data()),
        thrust::raw_pointer_cast(areas.data()));

    getRect<<<DIV_ROUND_UP(gs_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        W,
        H,
        gs_num,
        us.contiguous().data<float>(),
        thrust::raw_pointer_cast(areas.data()),
        depths.contiguous().data<float>(),
        grid,
        thrust::raw_pointer_cast(gs_rects.data()),
        thrust::raw_pointer_cast(gs_patch_num.data()));

    thrust::inclusive_scan(gs_patch_num.begin(), gs_patch_num.end(), gs_patch_offsets.begin());

    // patch_num: The total number of patches needs to be drawn
    uint patch_num = (uint)gs_patch_offsets[gs_num - 1];  // copy to cpu memory

    thrust::device_vector<uint64_t> patch_keys(patch_num);
    thrust::device_vector<uint> patch_gs_ids(patch_num);
    
    createKey<<<DIV_ROUND_UP(gs_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        gs_num,
        grid,
        thrust::raw_pointer_cast(gs_rects.data()),
        depths.contiguous().data<float>(),
        thrust::raw_pointer_cast(gs_patch_offsets.data()),
        thrust::raw_pointer_cast(patch_keys.data()),
        thrust::raw_pointer_cast(patch_gs_ids.data()));

    thrust::sort_by_key(patch_keys.begin(), patch_keys.end(), patch_gs_ids.begin());

    thrust::device_vector<uint2> gs_ranges(gs_num);

    getRange<<<DIV_ROUND_UP(patch_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        patch_num,
        thrust::raw_pointer_cast(patch_keys.data()),
        thrust::raw_pointer_cast(gs_ranges.data()));

    draw<<<grid, block>>>(
        W,
        H,
        thrust::raw_pointer_cast(gs_ranges.data()),
        thrust::raw_pointer_cast(patch_gs_ids.data()),
        us.contiguous().data<float>(),
        thrust::raw_pointer_cast(cov2d_inv.data()),
        alphas.contiguous().data<float>(),
        colors.contiguous().data<float>(),
        image.contiguous().data<float>());
    /*
    */
    return {image};
}
