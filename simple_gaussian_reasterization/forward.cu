#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <torch/extension.h>
#include "common.h"


inline __device__ void fetch2shared(
    int32_t n,
    const int2 range,
    const int *__restrict__ gs_id_per_patch,
    const float *__restrict__ us,
    const float *__restrict__ cov2d_inv,
    const float *__restrict__ alphas,
    const float *__restrict__ colors,
    float2 *shared_pos2d,
    float3 *shared_cinv2d,
    float *shared_alpha,
    float3 *shared_color)
{
    int i = blockDim.x * threadIdx.y + threadIdx.x;  // block idx
    int j = range.x + n * BLOCK_SIZE + i;  // patch idx
    if (j < range.y)
    {
        int gs_id = gs_id_per_patch[j];
        shared_pos2d[i].x = us[gs_id * 2];
        shared_pos2d[i].y = us[gs_id * 2 + 1];
        shared_cinv2d[i].x = cov2d_inv[gs_id * 3];
        shared_cinv2d[i].y = cov2d_inv[gs_id * 3 + 1];
        shared_cinv2d[i].z = cov2d_inv[gs_id * 3 + 2];
        shared_alpha[i] =   alphas[gs_id];
        shared_color[i].x = colors[gs_id * 3];
        shared_color[i].y = colors[gs_id * 3 + 1];
        shared_color[i].z = colors[gs_id * 3 + 2];
    }
}

__global__ void createKey(const int gs_num,
                          const dim3 grid,
                          const uint4 *__restrict__ rects,
                          const float *__restrict__ depths,
                          const uint *__restrict__ patch_offset_per_gs,
                          uint64_t *__restrict__ patch_keys,
                          int *__restrict__ gs_id_per_patch)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= gs_num)
		return;
    uint32_t off = (idx == 0) ? 0 : patch_offset_per_gs[idx - 1];
    uint4 rect = rects[idx];

		for (uint y = rect.y; y < rect.w; y++)
		{
			for (uint x = rect.x; x < rect.z; x++)
			{
				uint64_t key = (y * grid.x + x);
				key <<= 32;
                uint32_t depth_cm = depths[idx] * 1000; // mm
				key |= depth_cm;
				patch_keys[off] = key;
				gs_id_per_patch[off] = idx;
				off++;
			}
		}
}

__global__ void getRect(
    const int W,
    const int H,
    int gs_num,
    const float *__restrict__ us,
    const float *__restrict__ areas,
    const float *__restrict__ depths,
    const dim3 grid,
    uint4 *__restrict__ gs_rects,
    uint *__restrict__ patch_num_per_gs)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= gs_num)
		return;

    float d = depths[idx];
    float2 u = {us[idx*2], us[idx*2 + 1]};

    float x_norm =  u.x / W * 2.f - 1.f;
    float y_norm =  u.y / H * 2.f - 1.f;
    if (abs(x_norm) > 1.3 || abs(y_norm) > 1.3 || d < 0.1 || d > 100)
    {
        gs_rects[idx] = {0, 0, 0, 0};
        patch_num_per_gs[idx] = 0;
		return;
    }

    float xs = areas[idx*2];
    float ys = areas[idx*2 + 1];

    uint4 rect = {
		min(grid.x, max((int)0, (int)((u.x - xs) / BLOCK))),  // min_x
		min(grid.y, max((int)0, (int)((u.y - ys) / BLOCK))),  // min_y
        min(grid.x, max((int)0, (int)(DIV_ROUND_UP(u.x + xs, BLOCK)))),  // max_x
		min(grid.y, max((int)0, (int)(DIV_ROUND_UP(u.y + ys, BLOCK))))   // max_y
	};

    gs_rects[idx] = rect;
    patch_num_per_gs[idx] = (rect.z - rect.x) * (rect.w - rect.y);
}

__global__ void getOffset(
    const int patch_num,
    const uint64_t *__restrict__ patch_keys,
    int *__restrict__ patch_offset_per_tile)
{
    const int cur_patch = blockIdx.x * blockDim.x + threadIdx.x;

    if (cur_patch >= patch_num)
        return;

    const int prv_patch = cur_patch == 0 ? 0 : cur_patch - 1;

    uint32_t cur_tile = patch_keys[cur_patch] >> 32;
    uint32_t prv_tile = patch_keys[prv_patch] >> 32;

    if (cur_patch == 0)
    {
        patch_offset_per_tile[0] = 0;
        return;
    }

     if (cur_patch == patch_num - 1)
     {
        patch_offset_per_tile[cur_tile + 1] = patch_num;
     }

    if (prv_tile != cur_tile)
    {
        patch_offset_per_tile[cur_tile] = cur_patch;
    }
}

__global__ void  draw __launch_bounds__(BLOCK * BLOCK)(
    const int W,
    const int H,
    const int *__restrict__ patch_offset_per_tile,
    const int *__restrict__ gs_id_per_patch,
    const float *__restrict__ us,
    const float *__restrict__ cov2d_inv,
    const float *__restrict__ alphas,
    const float *__restrict__ colors,
    float *__restrict__ image,
    int *__restrict__ contrib,
    float *__restrict__ final_tau)

{
    const uint2 tile = {blockIdx.x, blockIdx.y};
    const uint2 pix = {tile.x * BLOCK + threadIdx.x,
                       tile.y * BLOCK + threadIdx.y};

    const int tile_idx = tile.y * gridDim.x + tile.x;
    const uint32_t pix_idx = W * pix.y + pix.x;

	const bool inside = pix.x < W && pix.y < H;
	const int2 range = {patch_offset_per_tile[tile_idx], patch_offset_per_tile[tile_idx + 1]};


	bool thread_is_finished = !inside;

	__shared__ float2 shared_pos2d[BLOCK_SIZE];
	__shared__ float3 shared_cinv2d[BLOCK_SIZE];
    __shared__ float  shared_alpha[BLOCK_SIZE];
    __shared__ float3 shared_color[BLOCK_SIZE];

	const int gs_num = range.y - range.x;

    float3 finial_color = {0, 0, 0};

    float tau = 1.0f;

    int cont = 0;

    // for all 2d gaussian 
    for (int i = 0; i < gs_num; i++)
    {
        int finished_thread_num = __syncthreads_count(thread_is_finished);

        if (finished_thread_num == BLOCK_SIZE)
            break;

        int j = i % BLOCK_SIZE;

        if (j == 0)
        {
            // fetch 2d gaussian data to share memory
            fetch2shared(i / BLOCK_SIZE,
                         range,
                         gs_id_per_patch,
                         us,
                         cov2d_inv,
                         alphas,
                         colors,
                         shared_pos2d,
                         shared_cinv2d,
                         shared_alpha,
                         shared_color);
            __syncthreads();
        }

        // get 2d gaussian info for current tile (pix share the same info within the tile)
        float2 u = shared_pos2d[j];
        float3 cinv = shared_cinv2d[j];
        float alpha = shared_alpha[j];
        float3 color = shared_color[j];
        float2 d = u - pix;

        // forward.md (5.1)
        // mahalanobis squared distance for 2d gaussian to this pix
        float maha_dist = max(0.0f,  mahaSqDist(cinv, d));

        float alpha_prime = min(0.99f, alpha * exp( -0.5f * maha_dist));

        if (alpha_prime < 0.002f)
            continue;

        // forward.md (5)
        finial_color +=  tau * alpha_prime * color;
        cont = cont + 1;  // how many gs contribute to this pixel. 

        // forward.md (5.2)
        float tau_new = tau * (1.f - alpha_prime);

        if (tau_new < 0.0001f)
        {
            thread_is_finished = true;
            continue;
        }
        tau = tau_new;
    }

    if (inside)
    {
        image[H * W * 0 + pix_idx] = finial_color.x;
        image[H * W * 1 + pix_idx] = finial_color.y;
        image[H * W * 2 + pix_idx] = finial_color.z;
        contrib[pix_idx] = cont;
        final_tau[pix_idx] = tau;
    }
}

__global__ void inverseCov2D(
    int gs_num,
    const float *__restrict__ cov2d,
    float *__restrict__ cov2d_inv,
    float *__restrict__ areas)
{
    // compute inverse of cov2d
    // Determine the drawing area of 2d Gaussian.

    const int gs_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (gs_id >= gs_num)
		return;
    // forward.md 5.3
    const float a = cov2d[gs_id * 3];
    const float b = cov2d[gs_id * 3 + 1];
    const float c = cov2d[gs_id * 3 + 2];

    const float det_inv = 1./(a*c - b*b);
    cov2d_inv[gs_id * 3 + 0] =  det_inv * c;
    cov2d_inv[gs_id * 3 + 1] = -det_inv * b;
    cov2d_inv[gs_id * 3 + 2] =  det_inv * a;
    areas[gs_id * 2 + 0] =  3 * sqrt(a);
    areas[gs_id * 2 + 1] =  3 * sqrt(c);
}

std::vector<torch::Tensor> forward(
    const int H,
    const int W,
    const torch::Tensor us,
    const torch::Tensor cov2d,
    const torch::Tensor alphas,
    const torch::Tensor depths,
    const torch::Tensor colors)
{
    auto float_opts = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat32);
    auto int_opts = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32);
    torch::Tensor image = torch::full({3, H, W}, 0.0, float_opts);
    torch::Tensor contrib = torch::full({H, W}, 0, int_opts);
    torch::Tensor final_tau = torch::full({H, W}, 0, float_opts);

    //gs:    2d gaussian;  a projection of a 3d gaussian onto a 2d image
    //tile:  a 16x16 area of 2d image
    //patch: a 2d gaussian may cover on many tiles, a 2d gaussian on a tile is called a patch

    // the total number of 2d gaussian.
    int gs_num = us.sizes()[0]; 
    
    dim3 grid(DIV_ROUND_UP(W, BLOCK), DIV_ROUND_UP(H, BLOCK), 1);
	dim3 block(BLOCK, BLOCK, 1);

    thrust::device_vector<uint4> gs_rects(gs_num);
    thrust::device_vector<uint>  patch_num_per_gs(gs_num);
    thrust::device_vector<uint>  patch_offset_per_gs(gs_num);
    // thrust::device_vector<float>  cov2d_inv(gs_num * 3);
    thrust::device_vector<float>  areas(gs_num * 2);

    torch::Tensor cov2d_inv = torch::full({gs_num * 3}, 0, float_opts);

    inverseCov2D<<<DIV_ROUND_UP(gs_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        gs_num,
        cov2d.contiguous().data_ptr<float>(),
        cov2d_inv.contiguous().data_ptr<float>(),
        thrust::raw_pointer_cast(areas.data()));
    hipDeviceSynchronize();

    getRect<<<DIV_ROUND_UP(gs_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        W,
        H,
        gs_num,
        us.contiguous().data_ptr<float>(),
        thrust::raw_pointer_cast(areas.data()),
        depths.contiguous().data_ptr<float>(),
        grid,
        thrust::raw_pointer_cast(gs_rects.data()),
        thrust::raw_pointer_cast(patch_num_per_gs.data()));
    hipDeviceSynchronize();

    thrust::inclusive_scan(patch_num_per_gs.begin(), patch_num_per_gs.end(), patch_offset_per_gs.begin());

    // patch_num: The total number of patches needs to be drawn
    uint patch_num = (uint)patch_offset_per_gs[gs_num - 1];  // copy to cpu memory

    thrust::device_vector<uint64_t> patch_keys(patch_num);
    thrust::device_vector<int> gs_id_per_patch(patch_num);
    
    createKey<<<DIV_ROUND_UP(gs_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        gs_num,
        grid,
        thrust::raw_pointer_cast(gs_rects.data()),
        depths.contiguous().data_ptr<float>(),
        thrust::raw_pointer_cast(patch_offset_per_gs.data()),
        thrust::raw_pointer_cast(patch_keys.data()),
        thrust::raw_pointer_cast(gs_id_per_patch.data()));
    hipDeviceSynchronize();

    thrust::sort_by_key(patch_keys.begin(), patch_keys.end(), gs_id_per_patch.begin());

    const uint tile_num = grid.x * grid.y;
    torch::Tensor patch_offset_per_tile = torch::full({tile_num+1}, 0, int_opts);

    getOffset<<<DIV_ROUND_UP(patch_num, BLOCK_SIZE), BLOCK_SIZE>>>(
        patch_num,
        thrust::raw_pointer_cast(patch_keys.data()),
        patch_offset_per_tile.contiguous().data_ptr<int>());
    hipDeviceSynchronize();

    draw<<<grid, block>>>(
        W,
        H,
        patch_offset_per_tile.contiguous().data_ptr<int>(),
        thrust::raw_pointer_cast(gs_id_per_patch.data()),
        us.contiguous().data_ptr<float>(),
        cov2d_inv.contiguous().data_ptr<float>(),
        alphas.contiguous().data_ptr<float>(),
        colors.contiguous().data_ptr<float>(),
        image.contiguous().data_ptr<float>(),
        contrib.contiguous().data_ptr<int>(),
        final_tau.contiguous().data_ptr<float>());
    hipDeviceSynchronize();

    torch::Tensor gsid_per_patch_torch = torch::from_blob(thrust::raw_pointer_cast(gs_id_per_patch.data()), 
        {static_cast<long>(gs_id_per_patch.size())}, torch::TensorOptions().dtype(torch::kInt32)).to(torch::kCUDA);

    return {image, contrib, final_tau, patch_offset_per_tile, gsid_per_patch_torch, cov2d_inv};
}
